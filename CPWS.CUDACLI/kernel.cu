#pragma unmanaged
#include "hip/hip_runtime.h"


#include <cmath>
#include <stdio.h>

using namespace std;
extern "C" float* Noise3DCuda(int x, int y, int z, float scale, float persistence, int iterations, int gpuid);

__device__ float FValues(float dim)
{
    if (dim == 3) return 0.3333333333333333f;
    return (sqrt(dim + 1) - 1) / dim;
}

__device__ float GValues(float dim)
{
    if (dim == 3) return 0.16666666666666666f;
    return ((dim + 1) - sqrt(dim + 1)) / ((dim + 1) * dim);
}

__device__ float Noise3DDevice(float* vals)
{
    float s = 0;
    float dimensions = 3;

    float vvals[3];
    float xvals[3];
    float ranks[3];
    int ivvals[3];

    for (int i = 0; i < dimensions; ++i)
    {
        s += vals[i];
    }
    s *= FValues(dimensions);

    float t = 0;
    for (int i = 0; i < dimensions; ++i)
    {
        vvals[i] = 0;
        xvals[i] = 0;
        ranks[i] = 0;
        t += ivvals[i] = (int)(vals[i] + s);
    }
    float G = 0;
    t *= G = GValues(dimensions);

    for (int i = dimensions - 1; i >= 0; --i)
    {
        xvals[i] = vals[i] - (ivvals[i] - t);
        for (int j = i + 1; j < dimensions; ++j)
            if (xvals[i] > xvals[j]) ranks[i]++; else ranks[j]++;
    }
    float n = 0;
    int temp = dimensions - 1;

    for (int i = 0; i < dimensions + 1; ++i)
    {
        t = 0.6;
        unsigned int hash = 98743247568;

        for (int j = 0; j < dimensions; ++j)
        {
            int ival = 0;
            if (i > 0) ival = (i == dimensions ? 1 : (ranks[j] >= temp ? 1 : 0));
            float vval = vvals[j] = i == 0 ? xvals[j] : xvals[j] - ival + i * G;

            t -= vval * vval;

            hash ^= (unsigned int)(1619 * (ivvals[j] + ival));
        }
        if (i > 0) temp--;
        if (t >= 0)
        {
            hash = hash * hash * hash * 60493;
            hash = (hash >> 13) ^ hash;

            hash &= 15;

            float result = 0.0;
            int current = 1;

            for (int j = dimensions - 1; j > -1; --j)
            {
                result += (hash & current) == 0 ? -vvals[j] : vvals[j];
                current *= 2;
            }

            n += (t * t) * t * t * result;
        }
    }

    return 32.0 * n;
}

__global__ void Noise3DOctaveKernel(float* scale, float* Persistence, int* height, int* width, int* dimZ, int* iterations, float* results)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    float maxAmp = 0;
    float cAmp = 1;
    float freq = *scale;
    float noise = 0;
    
    for (int i = 0; i < *iterations; i++)
    {
        float nX = x * freq;
        float nY = y * freq;
        float nZ = *dimZ * freq;

        float vvals[3] = { nX, nY, nZ };
        noise += Noise3DDevice(vvals) * cAmp;
        maxAmp += cAmp;
        cAmp *= *Persistence;
        freq *= 2;
    }

    results[y * *width + x] = noise / maxAmp;
}

extern "C" float* Noise3DCuda(int x, int y, int z, float scale, float persistence, int iterations, int gpuid)
{
    hipError_t cudaStatus;

    dim3 dimBlock(8, 8, 1);
    dim3 dimGrid(x / dimBlock.x, y / dimBlock.y, 1);

    float* dev_Scale = 0;
    float* dev_Persistence = 0;
    int* dev_Iterations = 0;
    int* dev_dimZ = 0;
    float* dev_Results = 0;
    int* dev_Height = 0;
    int* dev_Width = 0;
    float* results = (float*)malloc((x * y) * sizeof(float));;

    cudaStatus = hipSetDevice(gpuid);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "Setting CUDA Device failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_Scale, sizeof(float));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }
    cudaStatus = hipMalloc((void**)&dev_Persistence, sizeof(float));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }
    cudaStatus = hipMalloc((void**)&dev_Iterations, sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }
    cudaStatus = hipMalloc((void**)&dev_dimZ, sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }
    cudaStatus = hipMalloc((void**)&dev_Height, sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }
    cudaStatus = hipMalloc((void**)&dev_Width, sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMemcpy(dev_Scale, &scale, sizeof(float), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }
    cudaStatus = hipMemcpy(dev_Persistence, &persistence, sizeof(float), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }
    cudaStatus = hipMemcpy(dev_dimZ, &z, sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }
    cudaStatus = hipMemcpy(dev_Height, &y, sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }
    cudaStatus = hipMemcpy(dev_Width, &x, sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }
    cudaStatus = hipMemcpy(dev_Iterations, &iterations, sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_Results, (x * y) * sizeof(float));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    Noise3DOctaveKernel <<<dimGrid, dimBlock, 0 >>> (dev_Scale, dev_Persistence, dev_Height, dev_Width, dev_dimZ, dev_Iterations, dev_Results);

    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "Noise3DOctaveKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }

    // cudaDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
        goto Error;
    }

    // Copy output vector from GPU buffer to host memory.
    cudaStatus = hipMemcpy(results, dev_Results, (x * y) * sizeof(float), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

Error:
    hipFree(dev_Iterations);
    hipFree(dev_Persistence);
    hipFree(dev_Scale);
    hipFree(dev_dimZ);
    hipFree(dev_Width);
    hipFree(dev_Height);
    hipFree(dev_Results);

    return results;
}
#pragma managed