#include "hip/hip_runtime.h"
#pragma unmanaged
#include "hip/hip_runtime.h"
#include ""

#include <cmath>
#include <stdio.h>
#include "GpuTimer.h"

using namespace std;
extern "C" double* Noise3DCuda(short* source, int x, int y, int z, double scale, double persistence, int iterations, int gpuid);

static int* grad3 = new int[36]{ 
    1,1,0,
    -1,1,0,
    1,-1,0,
    -1,-1,0,
    1,0,1,
    -1,0,1,
    1,0,-1,
    -1,0,-1,
    0,1,1,
    0,-1,1,
    0,1,-1,
    0,-1,-1 
};

__device__ double FValues(double dim)
{
    return (sqrt(dim + 1) - 1) / dim;
}

__device__ double GValues(double dim)
{
    return ((dim + 1) - sqrt(dim + 1)) / ((dim + 1) * dim);
}

__device__ double Noise3DDevice(short* source, int *grad, double* dimX, double* dimY, double* dimZ)
{
    double n0, n1, n2, n3;
    double G = GValues(3);

    double s = (*dimX + *dimY + *dimZ) * FValues(3);

    int i = (int)floor(*dimX + s);
    int j = (int)floor(*dimY + s);
    int k = (int)floor(*dimZ + s);
    double t = (i + j + k) * G;

    double X0 = i - t;
    double Y0 = j - t;
    double Z0 = k - t;

    double x0 = *dimX - X0;
    double y0 = *dimY - Y0;
    double z0 = *dimZ - Z0;

    int i1, j1, k1;
    int i2, j2, k2;
    if (x0 >= y0)
    {
        if (y0 >= z0) { i1 = 1; j1 = 0; k1 = 0; i2 = 1; j2 = 1; k2 = 0; }
        else if (x0 >= z0) { i1 = 1; j1 = 0; k1 = 0; i2 = 1; j2 = 0; k2 = 1; }
        else { i1 = 0; j1 = 0; k1 = 1; i2 = 1; j2 = 0; k2 = 1; }
    }
    else
    { // x0<y0
        if (y0 < z0) { i1 = 0; j1 = 0; k1 = 1; i2 = 0; j2 = 1; k2 = 1; }
        else if (x0 < z0) { i1 = 0; j1 = 1; k1 = 0; i2 = 0; j2 = 1; k2 = 1; }
        else { i1 = 0; j1 = 1; k1 = 0; i2 = 1; j2 = 1; k2 = 0; }
    }

    double x1 = x0 - i1 + G;
    double y1 = y0 - j1 + G;
    double z1 = z0 - k1 + G;
    double x2 = x0 - i2 + 2.0 * G;
    double y2 = y0 - j2 + 2.0 * G;
    double z2 = z0 - k2 + 2.0 * G;
    double x3 = x0 - 1.0 + 3.0 * G;
    double y3 = y0 - 1.0 + 3.0 * G;
    double z3 = z0 - 1.0 + 3.0 * G;

    int ii = i & 255;
    int jj = j & 255;
    int kk = k & 255;

    int gi0 = (source[ii + source[jj + source[kk]]] % 12) * 3;
    int gi1 = (source[ii + i1 + source[jj + j1 + source[kk + k1]]] % 12) * 3;
    int gi2 = (source[ii + i2 + source[jj + j2 + source[kk + k2]]] % 12) * 3;
    int gi3 = (source[ii + 1 + source[jj + 1 + source[kk + 1]]] % 12) * 3;

    double t0 = 0.6 - x0 * x0 - y0 * y0 - z0 * z0;
    if (t0 < 0) n0 = 0.0;
    else
    {
        t0 *= t0;
        n0 = t0 * t0 * (grad[gi0] * x0 + grad[gi0 + 1] * y0 + grad[gi0 + 2] * z0);
    }
    double t1 = 0.6 - x1 * x1 - y1 * y1 - z1 * z1;
    if (t1 < 0) n1 = 0.0;
    else
    {
        t1 *= t1;
        n1 = t1 * t1 * (grad[gi1] * x1 + grad[gi1 + 1] * y1 + grad[gi1 + 2] * z1);
    }
    double t2 = 0.6 - x2 * x2 - y2 * y2 - z2 * z2;
    if (t2 < 0) n2 = 0.0;
    else
    {
        t2 *= t2;
        n2 = t2 * t2 * (grad[gi2] * x2 + grad[gi2 + 1] * y2 + grad[gi2 + 2] * z2);
    }
    double t3 = 0.6 - x3 * x3 - y3 * y3 - z3 * z3;
    if (t3 < 0) n3 = 0.0;
    else
    {
        t3 *= t3;
        n3 = t3 * t3 * (grad[gi3] * x3 + grad[gi3 + 1] * y3 + grad[gi3 + 2] * z3);
    }

    return 32.0 * (n0 + n1 + n2 + n3);
}

__global__ void Noise3DOctaveKernel(short* source, int* grad, double* scale, double* Persistence, int* height, int* width, int* dimZ, int* iterations, double* results)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    double maxAmp = 0;
    double cAmp = 1;
    double freq = *scale;
    double noise = 0;
    
    for (int i = 0; i < *iterations; i++)
    {
        double nX = x * freq;
        double nY = y * freq;
        double nZ = *dimZ * freq;

        noise += Noise3DDevice(source, grad, &nX, &nY, &nZ) * cAmp;
        maxAmp += cAmp;
        cAmp *= *Persistence;
        freq *= 2;
    }

    results[y * *width + x] = noise / maxAmp;
}

void cudaAllocate(void* dst, const void* src, size_t count, hipMemcpyKind kind) {
    if (hipMalloc((void**)dst, count) != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        throw - 1;
    }
    if (hipMemcpy(dst, src, count, kind) != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        throw - 1;
    }
}

extern "C" double* Noise3DCuda(short* source, int x, int y, int z, double scale, double persistence, int iterations, int gpuid)
{
    hipError_t cudaStatus;

    dim3 dimBlock(8, 8, 1);
    dim3 dimGrid(x / dimBlock.x, y / dimBlock.y, 1);

    double* dev_Scale = 0;
    double* dev_Persistence = 0;
    int* dev_Iterations = 0;
    int* dev_dimZ = 0;
    short* dev_Source = 0;
    int* dev_Grad = 0;
    double* dev_Results = 0;
    int* dev_Height = 0;
    int* dev_Width = 0;
    double* results = (double*)malloc((x * y) * sizeof(double));;

    cudaStatus = hipSetDevice(gpuid);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "Setting CUDA Device failed!");
        goto Error;
    }
    
    /*try {
        cudaAllocate(dev_Source, source, sizeof(short) * 512, hipMemcpyHostToDevice);
    }
    catch (int e) {
        goto Error;
    }*/
    cudaStatus = hipMalloc((void**)&dev_Source, sizeof(short) * 512);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }
    cudaStatus = hipMemcpy(dev_Source, source,  sizeof(short) * 512, hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }


    cudaStatus = hipMalloc((void**)&dev_Grad, sizeof(int) * 36);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }
    cudaStatus = hipMalloc((void**)&dev_Scale, sizeof(double));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }
    cudaStatus = hipMalloc((void**)&dev_Persistence, sizeof(double));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }
    cudaStatus = hipMalloc((void**)&dev_Iterations, sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }
    cudaStatus = hipMalloc((void**)&dev_dimZ, sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }
    cudaStatus = hipMalloc((void**)&dev_Height, sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }
    cudaStatus = hipMalloc((void**)&dev_Width, sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMemcpy(dev_Grad, grad3, sizeof(int) * 36, hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    cudaStatus = hipMemcpy(dev_Scale, &scale, sizeof(double), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }
    cudaStatus = hipMemcpy(dev_Persistence, &persistence, sizeof(double), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }
    cudaStatus = hipMemcpy(dev_dimZ, &z, sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }
    cudaStatus = hipMemcpy(dev_Height, &y, sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }
    cudaStatus = hipMemcpy(dev_Width, &x, sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }
    cudaStatus = hipMemcpy(dev_Iterations, &iterations, sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_Results, (x * y) * sizeof(double));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    Noise3DOctaveKernel << <dimGrid, dimBlock, 0 >> > (dev_Source, dev_Grad, dev_Scale, dev_Persistence, dev_Height, dev_Width, dev_dimZ, dev_Iterations, dev_Results);

    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "Noise3DOctaveKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }

    // hipDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
        goto Error;
    }

    // Copy output vector from GPU buffer to host memory.
    cudaStatus = hipMemcpy(results, dev_Results, (x * y) * sizeof(double), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

Error:
    hipFree(dev_Iterations);
    hipFree(dev_Persistence);
    hipFree(dev_Scale);
    hipFree(dev_dimZ);
    hipFree(dev_Source);
    hipFree(dev_Width);
    hipFree(dev_Height);
    hipFree(dev_Grad);
    hipFree(dev_Results);

    return results;
}
#pragma managed